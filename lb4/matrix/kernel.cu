#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <iostream>

#define BLOCK_SIZE 32

void getInfoCUDADevice(hipDeviceProp_t& prop, int id) {
    printf("CUDA device %i name  - %s\n", id, prop.name);
    printf("CUDA device %i Warp size in threads  - %i\n", id, prop.warpSize);
    printf("CUDA device %i Maximum number of threads per block  - %i\n", id, prop.maxThreadsPerBlock);
    printf("CUDA device %i multiprocessors count  - %i\n", id, prop.multiProcessorCount);
    printf("CUDA device %i Maximum size of each dimension of a block  - %i %i %i\n", id, prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("CUDA device %i Maximum size of each dimension of a grid  - %i %i %i\n", id, prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
}

__global__ void matrixMult(const int16_t *A, const int16_t *B, int16_t *result, int size) {
    int bx = blockIdx.x;  
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int ia = size * (gridDim.y * by + ty);
    int ib = gridDim.x * bx + tx;
    int ic = ia + ib;
    
    int16_t sum = 0;
    
    for (int k = 0; k < size; k++) {
        sum += A[ia + k] * B[k * size + ib];
    }
    result[ic] = sum;
}

void compareMatrix(const int16_t* f, const int16_t* s, int size) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            if (f[i * size + j] != s[i * size + j]) {
                std::cout << "Matrices are not equal\n";
                return;
            }
        }
    }
    std::cout << "Matrices are equal\n";
}

int main() {
    int count;
    hipDeviceProp_t prop;
    hipGetDeviceCount(&count);
    printf("Count CUDA devices - %i\n", count);
    hipGetDeviceProperties(&prop, count - 1);
    getInfoCUDADevice(prop, count - 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);    
    int size = 1024;
    for (int iter = 0; iter < 10; iter++) {
        std::cout << "Itteration number: " << iter + 1 << "\n";
        size_t byte_size = size * size * sizeof(int16_t);

        int16_t* h_A = (int16_t*)malloc(byte_size);
        int16_t* h_B = (int16_t*)malloc(byte_size);
        int16_t* h_C = (int16_t*)malloc(byte_size);
        int16_t* CPU_C = (int16_t*)malloc(byte_size);

        for (int i = 0; i < size * size; ++i) {
            h_A[i] = rand() % 100;
            h_B[i] = rand() % 100;
            CPU_C[i] = 0;
        }

        std::cout << "CPU: \n";
        hipEventRecord(start, 0);
        for (int i = 0; i < size; ++i) {
            for (int j = 0; j < size; ++j) {
                for (int k = 0; k < size; ++k) {
                    CPU_C[i * size + j] += h_A[i * size + k] * h_B[k * size + j];
                }
            }
        }
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float result_time_cpu;
        hipEventElapsedTime(&result_time_cpu, start, stop);
		std::cout << "Elapsed time: " << result_time_cpu << " ms\n";

        std::cout << "GPU: \n";

        int16_t* d_A = NULL;
        hipMalloc((void**)&d_A, byte_size);
        hipMemcpy(d_A, h_A, byte_size, hipMemcpyHostToDevice);

        int16_t* d_B = NULL;
        hipMalloc((void**)&d_B, byte_size);
        hipMemcpy(d_B, h_B, byte_size, hipMemcpyHostToDevice);

        int16_t* d_C = NULL;
        hipMalloc((void**)&d_C, byte_size);

        hipEventRecord(start, 0);

        const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        const dim3 grid(size / block.x, size / block.y);
        matrixMult <<< grid, block >>> (d_A, d_B, d_C, size);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float result_time_gpu;
        hipEventElapsedTime(&result_time_gpu, start, stop);
        std::cout << "Elapsed time: " << result_time_gpu << " ms\n";

        hipMemcpy(h_C, d_C, byte_size, hipMemcpyDeviceToHost);
        compareMatrix(h_C, CPU_C, size);

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        free(h_A);
        free(h_B);     
        free(h_C); 
        free(CPU_C);
    }
    hipEventDestroy(start);  
    hipEventDestroy(stop);

    return 0;
}
