#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/core/cuda_stream_accessor.hpp>
#include <iostream>
#include <chrono>
#include <string>

__global__ void modifyChannels(const uchar* src, uchar* blueChannel, uchar* yellowChannel, int rows, int cols) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        int idx = y * cols + x;
        uchar red = src[idx * 3 + 2];
        uchar green = src[idx * 3 + 1];
        uchar blue = src[idx * 3];

        // Ваши формулы для модификации каналов:
        blueChannel[idx] = blue - (green + blue) / 2;
        yellowChannel[idx] = red + green - 2 * (abs(red - green) + blue);
    }
}

int64_t mainLoop(cv::Mat& src, std::string &res, uint save)
{

    auto start = std::chrono::steady_clock::now();

    // Подготавливаем GPU матрицы
    cv::cuda::GpuMat gpuSrc, gpuBlueChannel, gpuYellowChannel;
    gpuSrc.upload(src);
    gpuBlueChannel.create(src.size(), CV_8UC1);
    gpuYellowChannel.create(src.size(), CV_8UC1);

    // Вычисляем размеры сетки и блока
    const dim3 block(16, 16);
    const dim3 grid((src.cols + block.x - 1) / block.x, (src.rows + block.y - 1) / block.y);

    // Запускаем ядро CUDA
    modifyChannels<<<grid, block>>>(gpuSrc.data, gpuBlueChannel.data, gpuYellowChannel.data, src.rows, src.cols);
    hipDeviceSynchronize();

    // Скачиваем результаты
    cv::Mat blueChannel, yellowChannel;
    gpuBlueChannel.download(blueChannel);
    gpuYellowChannel.download(yellowChannel);
    
    if (save)
    {
        cv::imwrite("../res/" + res + "_blue_channel.jpg", blueChannel);
        cv::imwrite("../res/" + res + "_yellow_channel.jpg", yellowChannel);
    }
    
    auto stop = std::chrono::steady_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop-start).count();

    return duration;
    
}

int main() 
{
    std::string input = "../img/16k.jpg";
    std::string output = "16k";
    int save = 1;

    cv::Mat src = cv::imread(input);
    if (src.empty()) {
        std::cerr << "Error loading the image" << std::endl;
        return -1;
    }
    
    auto start = std::chrono::steady_clock::now();

    for (int i = 0; i < 10; ++i)
    {
        int64_t duration = mainLoop(src, output, save);
        std::cout << "Picture 16k loop " << i+1 << " duration: " << duration << "mcs \n";

        if (i == 0)
        {
            save = 0;
        }
    }

    auto stop = std::chrono::steady_clock::now();

    std::cout << "16k pic 10 times duration: " << std::chrono::duration_cast<std::chrono::microseconds>(stop-start).count() << " mcs\n";
    std::cout << "\n";

    input = "../img/1.jpg";
    output = "1";
    save = 1;

    src = cv::imread(input);
    if (src.empty()) {
        std::cerr << "Error loading the image" << std::endl;
        return -1;
    }
    
    start = std::chrono::steady_clock::now();

    for (int i = 0; i < 10; ++i)
    {
        int64_t duration = mainLoop(src, output, save);
        std::cout << "Picture 1 loop " << i+1 << " duration: " << duration << " mcs\n";

        if (i == 0)
        {
            save = 0;
        }
    }

    stop = std::chrono::steady_clock::now();

    std::cout << "1st pic 10 times duration: " << std::chrono::duration_cast<std::chrono::microseconds>(stop-start).count() << " mcs\n";

    std::cout << "\n";

    input = "../img/2.jpg";
    output = "2";
    save = 1;

    src = cv::imread(input);
    if (src.empty()) {
        std::cerr << "Error loading the image" << std::endl;
        return -1;
    }
    
    start = std::chrono::steady_clock::now();

    for (int i = 0; i < 10; ++i)
    {
        int64_t duration = mainLoop(src, output, save);
        std::cout << "Picture 1 loop " << i+1 << " duration: " << duration << " mcs \n";

        if (i == 0)
        {
            save = 0;
        }
    }

    stop = std::chrono::steady_clock::now();

    std::cout << "2nd pic 10 times duration: " << std::chrono::duration_cast<std::chrono::microseconds>(stop-start).count() << " mcs\n";
    
    std::cout << "\n";

    input = "../img/3.jpg";
    output = "3";
    save = 1;

    src = cv::imread(input);
    if (src.empty()) {
        std::cerr << "Error loading the image" << std::endl;
        return -1;
    }
    
    start = std::chrono::steady_clock::now();

    for (int i = 0; i < 10; ++i)
    {
        int64_t duration = mainLoop(src, output, save);
        std::cout << "Picture 3 loop " << i+1 << " duration: " << duration << " mcs\n";

        if (i == 0)
        {
            save = 0;
        }
    }

    stop = std::chrono::steady_clock::now();

    std::cout << "3rd pic 10 times duration: " << std::chrono::duration_cast<std::chrono::microseconds>(stop-start).count() << " mcs\n";

    

    
}
